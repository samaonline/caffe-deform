#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/relu3_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void ReLU3Forward(const int n, const Dtype* in, Dtype* out,
    Dtype negative_slope, Dtype threshold_) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] > threshold_ ? in[index] : in[index] * negative_slope;
  }
}

template <typename Dtype>
void ReLU3Layer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  Dtype negative_slope = this->layer_param_.relu3_param().negative_slope();
  Dtype threshold_ = this->layer_param_.relu2_param().threshold();
  // NOLINT_NEXT_LINE(whitespace/operators)
  ReLU3Forward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data, negative_slope, threshold_);
  CUDA_POST_KERNEL_CHECK;
  // << " count: " << count << " bottom_data: "
  //     << (unsigned long)bottom_data
  //     << " top_data: " << (unsigned long)top_data
  //     << " blocks: " << CAFFE_GET_BLOCKS(count)
  //     << " threads: " << CAFFE_CUDA_NUM_THREADS;
}

template <typename Dtype>
__global__ void ReLU3Backward(const int n, const Dtype* in_diff,
    const Dtype* in_data, Dtype* out_diff, Dtype negative_slope,Dtype threshold_) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * ((in_data[index] > threshold_)
        + (in_data[index] <= threshold_) * negative_slope);
  }
}

template <typename Dtype>
void ReLU3Layer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    Dtype negative_slope = this->layer_param_.relu3_param().negative_slope();
    Dtype threshold_ = this->layer_param_.relu2_param().threshold();
    // NOLINT_NEXT_LINE(whitespace/operators)
    ReLU3Backward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, bottom_data, bottom_diff, negative_slope, threshold_);
    CUDA_POST_KERNEL_CHECK;
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(ReLU3Layer);


}  // namespace caffe
